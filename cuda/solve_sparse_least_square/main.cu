#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <stdlib.h>

#include <iostream>

#define CUDACHECK(call)              \
  {                                  \
    const hipError_t status = call; \
    assert(status == hipSuccess);   \
  }
#define CUSLVCHECK(call)                       \
  {                                            \
    const hipsolverStatus_t status = call;      \
    assert(status == HIPSOLVER_STATUS_SUCCESS); \
  }
#define CUSPCHECK(call)                        \
  {                                            \
    const hipsparseStatus_t status = call;      \
    assert(status == HIPSOLVER_STATUS_SUCCESS); \
  }

int main(int argc, char *argv[]) {
  hipsolverSpHandle_t cusolverH = NULL;
  csrqrInfo_t info = NULL;
  hipsparseMatDescr_t descrA = NULL;
  int *d_csrRowPtrA = NULL;
  int *d_csrColIndA = NULL;
  double *d_csrValA = NULL;
  double *d_b = NULL;
  double *d_x = NULL;

  /*     | 1.0    0    0    0 |
     A = |   0  2.0    0    0 |
         |   0    0  3.0    0 |
         | 0.1  0.1  0.1  4.0 | */
  const int m = 4;                                     // rows
  const int nnzA = 7;                                  // # of non-zero elements
  const int csrRowPtrA[m + 1] = {0, 1, 2, 3, 7};       // start of every row
  const int csrColIndA[nnzA] = {0, 1, 2, 0, 1, 2, 3};  // column index
  const double csrValA[nnzA] = {1.0, 2.0, 3.0, 0.1, 0.1, 0.1, 4.0};

  const double b[m] = {1.0, 1.0, 1.0, 1.0};

  CUSLVCHECK(hipsolverSpCreate(&cusolverH));
  CUSPCHECK(hipsparseCreateMatDescr(&descrA));
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);  // base-1
  CUSLVCHECK(cusolverSpCreateCsrqrInfo(&info));

  CUDACHECK(hipMalloc((void **)&d_csrValA, sizeof(double) * nnzA));
  CUDACHECK(hipMalloc((void **)&d_csrColIndA, sizeof(int) * nnzA));
  CUDACHECK(hipMalloc((void **)&d_csrRowPtrA, sizeof(int) * (m + 1)));
  CUDACHECK(hipMalloc((void **)&d_b, sizeof(double) * m));
  CUDACHECK(hipMalloc((void **)&d_x, sizeof(double) * m));

  CUDACHECK(hipMemcpy(d_csrValA, csrValA, sizeof(double) * nnzA, hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_csrColIndA, csrColIndA, sizeof(int) * nnzA, hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_csrRowPtrA, csrRowPtrA, sizeof(int) * (m + 1), hipMemcpyHostToDevice));
  CUDACHECK(hipMemcpy(d_b, b, sizeof(double) * m, hipMemcpyHostToDevice));

  double tol = 1e-6;
  int reorder = 1;
  int singularity;
  CUSLVCHECK(hipsolverSpDcsrlsvqr(cusolverH, m, nnzA, descrA, d_csrValA, d_csrRowPtrA,
                                       d_csrColIndA, d_b, tol, reorder, d_x, &singularity));

  double res_x[4];
  CUDACHECK(hipMemcpy(res_x, d_x, sizeof(double) * m, hipMemcpyDeviceToHost));
  for(int i = 0; i < 4; i++) {
    std::cout << res_x[i] << ", ";
  }
  std::cout << std::endl;

  return 0;
}

